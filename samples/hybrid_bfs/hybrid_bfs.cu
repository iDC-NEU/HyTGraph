#include "hip/hip_runtime.h"
// ----------------------------------------------------------------
// SEP-Graph: Finding Shortest Execution Paths for Graph Processing under a Hybrid Framework on GPU
// ----------------------------------------------------------------
// This source code is distributed under the terms of LICENSE
// in the root directory of this source distribution.
// ----------------------------------------------------------------
#include <functional>
#include <map>
//#define ARRAY_BITMAP
#include <framework/framework.cuh>
#include <framework/hybrid_policy.h>
#include <framework/clion_cuda.cuh>
#include <framework/variants/api.cuh>
#include <framework/common.h>
#include "hybrid_bfs_common.h"


DEFINE_int32(source_node, 0, "The source node for the BFS traversal (clamped to [0, nnodes-1])");
DEFINE_bool(sparse, false, "use async/push/dd + fusion for high-diameter");
DECLARE_bool(non_atomic);
DECLARE_int32(top_ranks);
DECLARE_bool(print_ranks);
DECLARE_string(output);
DECLARE_bool(check);

namespace hybrid_bfs
{
    template<typename TValue, typename TBuffer, typename TWeight, typename...UnusedData>
    struct BFS : sepgraph::api::AppBase<TValue, TBuffer, TWeight>
    {
        using sepgraph::api::AppBase<TValue, TBuffer, TWeight>::AccumulateBuffer;
        index_t m_source_node;
        bool m_non_atomic;

        BFS(index_t source, bool non_atomic) : m_source_node(source), m_non_atomic(non_atomic)
        {

        }

        __forceinline__ __device__

        TValue GetInitValue(index_t node) const override
        {
            return static_cast<TValue> (IDENTITY_ELEMENT);
        }

        __forceinline__ __device__

        TBuffer GetInitBuffer(index_t node) const override
        {
            TBuffer buffer;

            if (node == m_source_node)
            {
                buffer = 0;
            }
            else
            {
                buffer = UINT32_MAX;
            }

            return buffer;
        }

        __forceinline__ __host__
        __device__
        TBuffer

        GetIdentityElement() const override
        {
            return IDENTITY_ELEMENT;
        }

        __forceinline__ __device__

        utils::pair<TBuffer, bool> CombineValueBuffer(index_t node,
                                                      TValue *p_value,
                                                      TBuffer *p_buffer) override
        {
            TBuffer buffer = *p_buffer;
            bool schedule;

                schedule = false;

            if (*p_value > buffer)
            {
                *p_value = buffer;
                buffer += 1;
                schedule = true;
            }
            return utils::pair<TBuffer, bool>(buffer, schedule);
        }

        __forceinline__ __device__
        int AccumulateBuffer(index_t src,
                             index_t dst,
                             TBuffer *p_buffer,
                             TBuffer buffer) override
        {            
            atomicMin(p_buffer, buffer);    
            return 0;
        }
        
        __forceinline__ __device__
        TValue sum_value(index_t node, TValue value, TBuffer buffer) const override
        {
            if(value > buffer * 2)
                return TValue(2);

            return TValue(1);
        }

        __forceinline__ __device__

        bool IsActiveNode(index_t node, TBuffer buffer,TValue value) const override
        {
            //return buffer != IDENTITY_ELEMENT;
            return value > buffer;        
        }
    };
}

bool HybridBFS()
{
    LOG("HybridBFS\n");
    typedef sepgraph::engine::Engine<level_t, level_t, groute::graphs::NoWeight, hybrid_bfs::BFS, index_t, bool> HybridEngine;
    HybridEngine engine(sepgraph::policy::AlgoType::TRAVERSAL_SCHEME); //host_graph ready
    
    engine.LoadGraph();
    index_t source_node = min(max((index_t) 0, (index_t) FLAGS_source_node), engine.GetGraphDatum().nnodes - 1);//source_node not more than nnodes - 1
    
    sepgraph::common::EngineOptions engine_opt;
    auto regression = BFSHost(engine.CSRGraph(), source_node);

    
    engine.SetOptions(engine_opt);
    engine.InitGraph(source_node, FLAGS_non_atomic);
    engine.Start();
    engine.PrintInfo();

    utils::JsonWriter &writer = utils::JsonWriter::getInst();

    writer.write("non_atomic", FLAGS_non_atomic ? "YES" : "NO");

    const auto &levels = engine.GatherValue();
    bool success = true;

    if (FLAGS_check)
    {
        int errors = BFSCheckErrors(levels, regression);

        success = errors == 0;
        printf("total errors: %d\n", errors);
    }
    else
    {
        printf("Warning: Result not checked\n");
    }

    if (FLAGS_output.length() > 0)
    {
        BFSOutput(FLAGS_output.data(), levels);
    }
    return true;
}
