#include "hip/hip_runtime.h"
// ----------------------------------------------------------------
// SEP-Graph: Finding Shortest Execution Paths for Graph Processing under a Hybrid Framework on GPU
// ----------------------------------------------------------------
// This source code is distributed under the terms of LICENSE
// in the root directory of this source distribution.
// ----------------------------------------------------------------
#include <functional>
#include <map>
#include <framework/framework.cuh>
#include <framework/hybrid_policy.h>
#include <framework/clion_cuda.cuh>
#include <utils/cuda_utils.h>
#include "hybrid_sssp_common.h"
#include "../../include/groute/graphs/csr_graph.h"

DEFINE_int32(source_node,
             0, "The source node for the SSSP traversal (clamped to [0, nnodes-1])");
DEFINE_bool(sparse,
            false, "use async/push/dd + fusion for high-diameter");
DECLARE_int32(top_ranks);
DECLARE_bool(print_ranks);
DECLARE_string(output);
DECLARE_bool(check);
DECLARE_int32(prio_delta);

namespace hybrid_sssp
{
    template<typename TValue, typename TBuffer, typename TWeight, typename...UnusedData>
    struct SSSP : sepgraph::api::AppBase<TValue, TBuffer, TWeight>
    {
        using sepgraph::api::AppBase<TValue, TBuffer, TWeight>::AccumulateBuffer;
        index_t m_source_node;

        SSSP(index_t source_node) : m_source_node(source_node)
        {
         
        }

        __forceinline__ __device__

        TValue GetInitValue(index_t node) const override
        {
            return static_cast<TValue> (IDENTITY_ELEMENT);
        }

        __forceinline__ __device__

        TBuffer GetInitBuffer(index_t node) const override 
        {
            TBuffer buffer;
            if (node == m_source_node)//source_node = 1
            {
                buffer = 0;
            }
            else
            {
                buffer = IDENTITY_ELEMENT;
            }
            return buffer;
        }

        __forceinline__ __host__ __device__
        TBuffer GetIdentityElement() const override
        {
            return IDENTITY_ELEMENT;
        }

        __forceinline__ __device__
        utils::pair<TBuffer, bool> CombineValueBuffer(index_t node,
                                                      TValue *p_value,
                                                      TBuffer *p_buffer) override
        {

            //TBuffer buffer = atomicExch(p_buffer, IDENTITY_ELEMENT);
            TBuffer buffer = *p_buffer;
            bool schedule = false;

            if (*p_value > buffer)
            {
                *p_value = buffer;
                schedule = true;
            }
            return utils::pair<TBuffer, bool>(buffer, schedule);
        }

        __forceinline__ __device__
        int AccumulateBuffer(index_t src,
                             index_t dst,
                             TWeight weight,
                             TBuffer *p_buffer,    //dst_buffer
                             TBuffer buffer) override   //src_buffer
        {
            //if (*p_buffer > weight + buffer)
                atomicMin(p_buffer, buffer + weight); // calculate min of left and right and put min in left and return 

            return 1;
        }

        __forceinline__ __device__

        bool IsActiveNode(index_t node, TBuffer buffer,TValue value) const override
        {
            return buffer < value;
        }
        
        __forceinline__ __device__
        TValue sum_value(index_t node, TValue value,TBuffer buffer) const override
        {
            if(value > buffer * 2)
                return TValue(2);

            return TValue(1);
        }


        __forceinline__ __device__

        bool IsHighPriority(TBuffer current_priority, TBuffer buffer) const override
        {
            return current_priority > buffer;
        }
    };
}


/**
 * Δ = cw/d,
    where d is the average degree in the graph, w is the average
    edge weight, and c is the warp width (32 on our GPUs).
 * @return
 */

bool HybridSSSP()
{
    assert(UINT32_MAX == UINT_MAX);
    typedef sepgraph::engine::Engine<distance_t, distance_t, distance_t, hybrid_sssp::SSSP, index_t> HybridEngine;
    HybridEngine engine(sepgraph::policy::AlgoType::TRAVERSAL_SCHEME);
    engine.LoadGraph();

    index_t source_node = min(max((index_t) 0, (index_t) FLAGS_source_node), engine.GetGraphDatum().nnodes - 1);

    sepgraph::common::EngineOptions engine_opt;


    groute::graphs::host::CSRGraph csr_graph = engine.CSRGraph();
    double weight_sum = 0;
    for (uint64_t edge = 0; edge < csr_graph.nedges; edge++)
    {
        weight_sum += csr_graph.edge_weights[edge];
    }

    /**
     * We select a similar heuristic, Δ = cw/d,
        where d is the average degree in the graph, w is the average
        edge weight, and c is the warp width (32 on our GPUs)
        Link: https://people.csail.mith.edu/jshun/papers/DBGO14.pdf
     */
    int init_prio = 32 * (weight_sum / csr_graph.nedges) /
                    (1.0 * csr_graph.nedges / csr_graph.nnodes);

    printf("Priority delta: %u\n", init_prio);

    if (FLAGS_sparse)
    {
        engine_opt.SetFused();
        engine_opt.SetTwoLevelBasedPriority(init_prio);
        engine_opt.ForceVariant(sepgraph::common::AlgoVariant::ASYNC_PUSH_DD);
        engine_opt.SetLoadBalancing(sepgraph::common::MsgPassing::PUSH, sepgraph::common::LoadBalancing::NONE);
    }

    if (FLAGS_prio_delta > 0)
    {
        printf("Enable priority for scale-free dataset\n");
        engine_opt.SetTwoLevelBasedPriority(FLAGS_prio_delta);
    }

    engine.SetOptions(engine_opt);
    engine.InitGraph(source_node);
    engine.Start(init_prio);
    engine.PrintInfo();

    const auto &distances = engine.GatherValue();
    const auto *p_weight_datum =
            const_cast<sepgraph::graphs::GraphDatum<distance_t, distance_t, distance_t> &>(engine.GetGraphDatum()).m_csr_edge_weight_datum.GetHostDataPtr();

    bool success = true;
    if (FLAGS_check)
    {
        auto regression = SSSPHostNaive(engine.CSRGraph(), p_weight_datum, source_node);
        int errors = SSSPCheckErrors(distances, regression);

        success = errors == 0;
        printf("total errors: %d\n", errors);
    }
    else
    {
        printf("Warning: Result not checked\n");
    }

    if (FLAGS_output.length() > 0)
    {
        SSSPOutput(FLAGS_output.data(), distances);
    }
    return success;
}