#include "hip/hip_runtime.h"
// ----------------------------------------------------------------
// SEP-Graph: Finding Shortest Execution Paths for Graph Processing under a Hybrid Framework on GPU
// ----------------------------------------------------------------
// This source code is distributed under the terms of LICENSE
// in the root directory of this source distribution.
// ----------------------------------------------------------------
#include <framework/framework.cuh>
#include <framework/hybrid_policy.h>
#include <framework/clion_cuda.cuh>
#include <framework/variants/api.cuh>
#include "hybrid_pr_common.h"
#include <functional>
#include <map>

// Priority
DEFINE_double(cut_threshold, 0, "Cut threshold for index calculation");
DEFINE_bool(sparse, false, "disable load-balancing for sparse graph");
DECLARE_double(error);
DECLARE_int32(top_ranks);
DECLARE_bool(print_ranks);
DECLARE_string(output);

namespace hybrid_pr
{
    template<typename TValue, typename TBuffer, typename TWeight, typename...UnusedData>
    struct PageRank : sepgraph::api::AppBase<TValue, TBuffer, TWeight>
    {

        /*
         * For get rid of compiler bug: It's strange that if base class has virtual function, we must add a member for subclass.
         *
         * Error: Internal Compiler Error (codegen): "there was an error in verifying the lgenfe output!"
         */
        double m_error;
        using sepgraph::api::AppBase<TValue, TBuffer, TWeight>::AccumulateBuffer;

        PageRank(double error) : m_error(error)
        {
	      
        }

        __forceinline__ __device__

        TValue GetInitValue(index_t node) const override
        {
            return 0.0f;
        }

        __forceinline__ __device__

        TBuffer GetInitBuffer(index_t node) const override
        {
            return 1 - ALPHA;
        }

        __forceinline__ __host__
        __device__
                TBuffer

        GetIdentityElement() const override
        {
            return 0.0f;
        }

        __forceinline__ __device__

        utils::pair<TBuffer, bool> CombineValueBuffer(index_t node,
                                                      TValue *p_value,
                                                      TBuffer *p_buffer) override
        {
            TBuffer buffer = atomicExch(p_buffer, IDENTITY_ELEMENT);
            bool schedule = false;
            if (buffer > 0.01)
            {
                schedule = true;
                *p_value += buffer;

                int out_degree = this->m_csr_graph.end_edge(node) -
                                     this->m_csr_graph.begin_edge(node);
                buffer = ALPHA * buffer / out_degree;

            }

            return utils::pair<TBuffer, bool>(buffer, schedule);
        }

        __forceinline__ __device__

        int AccumulateBuffer(index_t src,
                             index_t dst,
                             TBuffer *p_buffer,
                             TBuffer buffer) override
        {
            atomicAdd(p_buffer, buffer);
            return 0;
        }

        __forceinline__ __device__

        bool IsActiveNode(index_t node, TBuffer buffer, TValue value) const override
        {
            return buffer > m_error;
        }
        
        __forceinline__ __device__

        TValue sum_value(index_t node, TValue value, TBuffer buffer) const override
        {
            return buffer;
        }

        __forceinline__ __device__

        bool IsHighPriority(TBuffer current_priority, TBuffer buffer) const override
        {
            return current_priority <= buffer;
        }
    };
}

bool HybridPageRank()
{
    LOG("HybridPageRank\n");
    typedef sepgraph::engine::Engine<rank_t, rank_t, groute::graphs::NoWeight, hybrid_pr::PageRank, double> HybridEngine;
    HybridEngine engine(sepgraph::policy::AlgoType::ITERATIVE_SCHEME);
    sepgraph::engine::EngineOptions engine_opt;


    engine.SetOptions(engine_opt);
    engine.LoadGraph();
    engine.InitGraph(FLAGS_error);
    engine.Start();
    engine.PrintInfo();
    utils::JsonWriter &writer = utils::JsonWriter::getInst();

    writer.write("error_tolerance", (float) FLAGS_error);

    const std::vector<rank_t> &ranks = engine.GatherValue();
    const std::vector<rank_t> &residual = engine.GatherBuffer();
    rank_t max_residual = *std::max_element(residual.begin(), residual.end());

    double pr_sum = 0;

    for (rank_t rank:ranks)
    {
        pr_sum += rank;
    }

    //printf("Total rank : %f\n", pr_sum);
    //printf("max residual: %f\n", max_residual);

    bool success = true;
    if (FLAGS_check)
    {
        auto regression = PageRankHost(engine.CSRGraph());
        auto gathered_output = engine.GatherValue();
        int errors = PageRankCheckErrors(gathered_output, regression);

        success = errors == 0;
        printf("total errors: %d\n", errors);
    }
    else
    {
        printf("Warning: Result not checked\n");
    }

    if (FLAGS_output.length() > 0)
    {
        PageRankOutput(FLAGS_output.data(), ranks);
    }

    return success;
}